#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "bmp/EasyBMP.h"

#define PI 3.14159265358979
#define WINDOW_SIZE 3
#define WINDOW_LENGHT WINDOW_SIZE *WINDOW_SIZE

using namespace std;

texture<float, hipTextureType2D, hipReadModeElementType> tex;

__global__ void cudaBilateral(float *output, int imageWidth, int imageHeight)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (!(row < imageHeight && col < imageWidth)) {
        return;
    }

    float filter[WINDOW_LENGHT] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
    for (int x = 0; x < WINDOW_SIZE; x++)
    {
        for (int y = 0; y < WINDOW_SIZE; y++)
        {
            filter[x * WINDOW_SIZE + y] = tex2D(tex, col + y - 1, row + x - 1);
        }
    }
    for (int i = 0; i < WINDOW_LENGHT; i++)
    {
        for (int j = i + 1; j < WINDOW_LENGHT; j++)
        {
            
        }
    }
    output[row * imageWidth + col] = filter[(int)(WINDOW_LENGHT / 2)];
}

float *readLikeGrayScale(char *filePathInput, unsigned int *rows, unsigned int *cols)
{
    BMP Input;
    Input.ReadFromFile(filePathInput);
    *rows = Input.TellHeight();
    *cols = Input.TellWidth();
    float *grayscale = (float *)calloc(*rows * *cols, sizeof(float));
    for (int j = 0; j < *rows; j++)
    {
        for (int i = 0; i < *cols; i++)
        {
            float gray = (float)floor(0.299 * Input(i, j)->Red +
                                      0.587 * Input(i, j)->Green +
                                      0.114 * Input(i, j)->Blue);
            grayscale[j * *cols + i] = gray;
        }
    }
    return grayscale;
}

void writeImage(char *filePath, float *grayscale, unsigned int rows, unsigned int cols)
{
    BMP Output;
    Output.SetSize(cols, rows);
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            RGBApixel pixel;
            pixel.Red = grayscale[i * cols + j];
            pixel.Green = grayscale[i * cols + j];
            pixel.Blue = grayscale[i * cols + j];
            pixel.Alpha = 0;
            Output.SetPixel(j, i, pixel);
        }
    }
    Output.WriteToFile(filePath);
}


float g(int x, int y, int i, int j) {
    return float(sqrt(pow(x - i, 2) + pow(y - j, 2)));
}

float f(float x, float sigma) {
    return exp(-(pow(x, 2))/(2 * pow(sigma, 2))) / (2 * PI * pow(sigma, 2));
}

float r(float x, float x1, float sigma) {
    return exp(pow(f(x, sigma) - f(x1, sigma), 2) / sigma);
}

void hostBilateral(float* output, unsigned int rows, unsigned int cols) {
    float sigma = 16.0;
    cout << rows<< endl;
    cout << cols << endl;
    for (int i = 0; i<rows; i++) {
        for (int j = 0; j<cols; j++){
                int h =0, k = 0;
                for (int x = -1; x<WINDOW_SIZE-1; x++) {
                    for (int y = -1; y< WINDOW_SIZE-1; y++) {
                        int currentRow = i + x * rows;
                        int currentColumn = j+y;
                        if (currentRow < 0 || currentRow > rows) {
                            currentRow = i-x;
                        }
                        if (currentColumn < 0 || currentColumn > rows) {
                            currentColumn = j - y;
                        }
                        
                        h += f(output[currentRow + currentColumn], sigma) + g(x, y, i, j) + r(output[i + j], output[currentRow +currentRow], sigma);
                        k += g(x, y, i, j) + r(output[i + j], output[currentRow + currentColumn], sigma);
                    }
                }
                h /= k;
                // cout << output[i*rows+j] << endl;
                cout << h << endl;
                output[i*rows+j] = h;
        }
    }
    
}

int main()
{
    float *grayscale = 0;
    unsigned int rows, cols;

    grayscale = readLikeGrayScale("lena.bmp", &rows, &cols);
    hostBilateral(grayscale, rows, cols);
    writeImage("afterRead.bmp", grayscale, rows, cols);
    // hipChannelFormatDesc channelDesc =
    //     hipCreateChannelDesc(32, 0, 0, 0,
    //                           hipChannelFormatKindFloat);
    // hipArray *cuArray;
    // hipMallocArray(&cuArray, &channelDesc, cols, rows);

    // hipMemcpyToArray(cuArray, 0, 0, grayscale, rows * cols * sizeof(float),
    //                                   hipMemcpyHostToDevice);

    // tex.addressMode[0] = hipAddressModeWrap;
    // tex.addressMode[1] = hipAddressModeWrap;
    // tex.filterMode = hipFilterModeLinear;

    // hipBindTextureToArray(tex, cuArray, channelDesc));

    // float *dev_output, *output;
    // output = (float *)calloc(rows * cols, sizeof(float));
    // hipMalloc(&dev_output, rows * cols * sizeof(float));

    // dim3 dimBlock(16, 16);
    // dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
    //              (rows + dimBlock.y - 1) / dimBlock.y);
    // cudaBilateral<<<dimGrid, dimBlock>>>(dev_output, cols, rows);
    // hipMemcpy(output, dev_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // writeImage("result.bmp", output, rows, cols);
    // hipFreeArray(cuArray);
    // hipFree(dev_output);
    return 0;
}

